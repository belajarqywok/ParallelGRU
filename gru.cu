#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

using namespace std;

extern float toBW(int bytes, float sec);

__global__ void
copy_data_kernel(float* x_t, int x_height, int x_width, float* data, int m, int n, int start_i, int j) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = index / x_width;
    int index_x = index % x_width;
    if (index < x_height * x_width) {
        x_t[index] = data[(start_i + index_y) * n + j + index_x];
    }
}

__global__ void
mat_init_zeros_kernel(float* a, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        a[index] = 0.0;
    }
}

__global__ void
mat_copy_kernel(float* dest, float* src, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        dest[index] = src[index];
    }
}

__global__ void
mat_multiplication_kernel(float* a, float* b, float* c, int c_width, int c_height, int a_width) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = index / c_width;
    int index_x = index % c_width;
    if (index < c_width * c_height) {
        float tmp = 0;
        for (int i = 0; i < a_width; ++i) {
            tmp += a[index_y * a_width + i] * b[i * c_width + index_x];
        }
        c[index_y * c_width + index_x] = tmp;
    }
}

__global__ void 
mat_add_kernel(float* a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] + b[index];
}

__global__ void 
mat_add_b_kernel(float* a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] + b[index % width];
}

__global__ void 
mat_one_sub_kernel(float* a, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = 1 - a[index];
}

__global__ void 
mat_hadamard_kernel(float*a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] * b[index];
}

__global__ void 
mat_sigmoid_kernel(float* a, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        a[index] = 1 / (1 + exp(-1 * a[index]));
}

__global__ void 
mat_tanh_kernel(float* a, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        a[index] = tanh(a[index]);
}

// x_t: width: 28, height: batch_size
// old_h_t: width: hidden_unit, height: batch_size
// new_h_t: width: hidden_unit, height: batch_size
// w_z, w_r, w_h: width: hidden_unit, height: 28
// u_z, u_r, u_h: width: hidden_unit, height: hidden_unit
// b_z, b_r, b_h: width: hidden_unit, height: 1
void gru_forward_kernel(int batch_size, int x_width, int hidden_unit,
                float* x_t, float* old_h_t, float* new_h_t,
                float* w_z, float* w_r, float* w_h,
                float* u_z, float* u_r, float* u_h,
                float* b_z, float* b_r, float* b_h) {

    const int threadsPerBlock = 512;
    const int blocks = (hidden_unit * batch_size + threadsPerBlock - 1) / threadsPerBlock;

    float* tmp1;
    float* tmp2;
    hipMalloc((void **)&tmp1, hidden_unit * batch_size * sizeof(float));
    hipMalloc((void **)&tmp2, hidden_unit * batch_size * sizeof(float));

    // z_t = sigmoid(x_t * w_z + old_h_t * u_z + b_z)
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(x_t, w_z, tmp1, hidden_unit, batch_size, x_width);
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(old_h_t, u_z, tmp2, hidden_unit, batch_size, hidden_unit);

    float* z_t;
    hipMalloc((void **)&z_t, hidden_unit * batch_size * sizeof(float));
    mat_add_kernel<<<blocks, threadsPerBlock>>>(tmp1, tmp2, z_t, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(z_t, b_z, z_t, hidden_unit, batch_size);
    mat_sigmoid_kernel<<<blocks, threadsPerBlock>>>(z_t, hidden_unit, batch_size);

    // r_t = sigmoid(x_t * w_r + old_h_t * u_r + b_r)
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(x_t, w_r, tmp1, hidden_unit, batch_size, x_width);
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(old_h_t, u_r, tmp2, hidden_unit, batch_size, hidden_unit);
    
    float* r_t;
    hipMalloc((void **)&r_t, hidden_unit * batch_size * sizeof(float));

    mat_add_kernel<<<blocks, threadsPerBlock>>>(tmp1, tmp2, r_t, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(r_t, b_r, r_t, hidden_unit, batch_size);
    mat_sigmoid_kernel<<<blocks, threadsPerBlock>>>(r_t, hidden_unit, batch_size);

    // h_hat = phi(x_t * w_h + (r_t . old_h_t) * u_h + b_h)
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(x_t, w_h, tmp1, hidden_unit, batch_size, x_width);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(r_t, old_h_t, r_t, hidden_unit, batch_size);
    mat_multiplication_kernel<<<blocks, threadsPerBlock>>>(r_t, u_h, tmp2, hidden_unit, batch_size, hidden_unit);

    float* h_hat;
    hipMalloc((void **)&h_hat, hidden_unit * batch_size * sizeof(float));

    mat_add_kernel<<<blocks, threadsPerBlock>>>(tmp1, tmp2, h_hat, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(h_hat, b_h, h_hat, hidden_unit, batch_size);
    mat_tanh_kernel<<<blocks, threadsPerBlock>>>(h_hat, hidden_unit, batch_size);

    // new_h_t = (1-z_t).old_h_t + z_t.h_hat
    float* tmp3;
    hipMalloc((void **)&tmp3, hidden_unit * batch_size * sizeof(float));
    mat_one_sub_kernel<<<blocks, threadsPerBlock>>>(z_t, tmp3, hidden_unit, batch_size);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(tmp3, old_h_t, tmp3, hidden_unit, batch_size);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(z_t, h_hat, h_hat, hidden_unit, batch_size);
    mat_add_kernel<<<blocks, threadsPerBlock>>>(tmp3, h_hat, new_h_t, hidden_unit, batch_size);

}

void one_iteration_cuda(int num_data, int batch_size, int window_size, int x_width, int hidden_unit,
                        float* old_h_t, float* new_h_t,
                        float* w_z, float* w_r, float* w_h,
                        float* u_z, float* u_r, float* u_h,
                        float* b_z, float* b_r, float* b_h,
                        float* dense, float* predict, float* arr_data, int m, int n) {

    double startTime = CycleTimer::currentSeconds();

    // allocate variables
    float *device_data;

    float *device_x_t;
    float *device_old_h_t;
    float *device_new_h_t;
    float *device_w_z;
    float *device_w_r;
    float *device_w_h;
    float *device_u_z;
    float *device_u_r;
    float *device_u_h;
    float *device_b_z;
    float *device_b_r;
    float *device_b_h;

    float *device_dense;
    float *device_predict;

    hipMalloc((void**)&device_data, m * n * sizeof(float));

    hipMalloc((void **)&device_x_t, batch_size * x_width * sizeof(float));
    hipMalloc((void **)&device_old_h_t, batch_size * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_new_h_t, batch_size * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_z, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_h, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_r, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_z, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_h, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_r, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_z, 1 * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_h, 1 * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_r, 1 * hidden_unit * sizeof(float));

    hipMalloc((void **)&device_dense, hidden_unit * 1 * sizeof(float));
    hipMalloc((void **)&device_predict, batch_size * 1 * sizeof(float));

    hipMemcpy(device_data, arr_data, m * n * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(device_old_h_t, old_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_new_h_t, new_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_z, w_z, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_h, w_h, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_r, w_r, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_z, u_z, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_h, u_h, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_r, u_r, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_z, b_z, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_h, b_h, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_r, b_r, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_dense, dense, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_predict, predict, 1 * batch_size * sizeof(float), hipMemcpyHostToDevice);

    const int threadsPerBlock = 512;
    const int blocks_h = (hidden_unit * batch_size + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_predict = (batch_size + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks_x = (x_width * batch_size + threadsPerBlock - 1) / threadsPerBlock;

    double iterStartTime = CycleTimer::currentSeconds();
    // One iteration, loop through all data point
    for (int i = 0; i < num_data; i += batch_size) {

        // batch_size * (num_data * x_width)
        int start_i = i;
        int end_i = min(num_data, i + batch_size);
        int batch = end_i - start_i;

        // for each time step
        
        for (int j = 0; j < window_size; j++) {

            copy_data_kernel<<<blocks_x, threadsPerBlock>>>(device_x_t, batch, x_width, device_data, m, n, start_i, j);

            // one forward iteration: 
            gru_forward_kernel(batch_size, x_width, hidden_unit, device_x_t, device_old_h_t, device_new_h_t, 
                device_w_z, device_w_r, device_w_h, device_u_z, device_u_r, device_u_h, device_b_z, device_b_r, device_b_h); 
        
            mat_copy_kernel<<<blocks_h, threadsPerBlock>>>(device_old_h_t, device_new_h_t, batch_size * hidden_unit);
            mat_init_zeros_kernel<<<blocks_h, threadsPerBlock>>>(device_new_h_t, batch_size * hidden_unit);

        }

        // inference
        mat_multiplication_kernel<<<blocks_predict, threadsPerBlock>>>(device_dense, device_old_h_t, device_predict, batch_size, 1, hidden_unit);

        hipMemcpy(predict, device_predict, batch_size * sizeof(float), hipMemcpyDeviceToHost);
        // if (i == 0) {
        //     for (int k = 0; k < batch_size; k++) {
        //         printf("%.3f ", predict[k]);
        //     }
        //     printf("\n");
        // }
        
        // calculate loss
        // gru_backward
        // update variables
        
    }
    double iterEndTime = CycleTimer::currentSeconds();

    hipFree(device_x_t);
    hipFree(device_old_h_t);
    hipFree(device_new_h_t);
    hipFree(device_w_z);
    hipFree(device_w_h);
    hipFree(device_w_r);
    hipFree(device_u_z);
    hipFree(device_u_h);
    hipFree(device_u_r);
    hipFree(device_b_z);
    hipFree(device_b_h);
    hipFree(device_b_r);
    hipFree(device_dense);
    hipFree(device_predict);

    double endTime = CycleTimer::currentSeconds();
    printf("GPU Overall: %.3f ms\n", 1000.f * (endTime - startTime));
    printf("GPU Compute: %.3f ms\n", 1000.f * (iterEndTime - iterStartTime));
    
}


void
print_cuda_info() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
