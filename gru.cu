#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

void mat_multiplication_kernel(float* a, float* b, float* c, int c_width, int c_height, int a_width) {
    for (int i = 0; i < c_height; ++i) {
        for (int j = 0; j < c_width; ++j) {
            for (int k = 0; k < a_width; ++k) {
                c[i * c_width + j] += a[i * a_width + k] * b[k * c_width + j];
            }
        }
    }
}

__global__ void 
mat_add_kernel(float* a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] + b[index];
}

__global__ void 
mat_add_b_kernel(float* a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] + b[index % width];
}

__global__ void 
mat_one_sub_kernel(float* a, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = 1 - a[index];
}

__global__ void 
mat_hadamard_kernel(float*a, float* b, float* res, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        res[index] = a[index] * b[index];
}

__global__ void 
mat_sigmoid_kernel(float* a, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        a[index] = 1 / (1 + exp(-1 * a[index]));
}

__global__ void 
mat_tanh_kernel(float* a, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < width * height) 
        a[index] = tanh(a[index]);
}

// x_t: width: 28, height: batch_size
// old_h_t: width: hidden_unit, height: batch_size
// new_h_t: width: hidden_unit, height: batch_size
// w_z, w_r, w_h: width: hidden_unit, height: 28
// u_z, u_r, u_h: width: hidden_unit, height: hidden_unit
// b_z, b_r, b_h: width: hidden_unit, height: 1
void gru_forward_kernel(int batch_size, int x_width, int hidden_unit,
                float* x_t, float* old_h_t, float* new_h_t,
                float* w_z, float* w_r, float* w_h,
                float* u_z, float* u_r, float* u_h,
                float* b_z, float* b_r, float* b_h) {
    
    // remove after paralleled matrix multiplication
    float* host_x_t = (float*)malloc(batch_size * x_width * sizeof(float));
    hipMemcpy(host_x_t, x_t, batch_size * x_width * sizeof(float), hipMemcpyDeviceToHost);
    float* host_w_z = (float*)malloc(hidden_unit * x_width * sizeof(float));
    hipMemcpy(host_w_z, w_z, hidden_unit * x_width * sizeof(float), hipMemcpyDeviceToHost);
    float* host_old_h_t = (float*)malloc(batch_size * hidden_unit * sizeof(float));
    hipMemcpy(host_old_h_t, old_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);
    float* host_u_z = (float*)malloc(hidden_unit * hidden_unit * sizeof(float));
    hipMemcpy(host_u_z, u_z, hidden_unit * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);
    float* host_w_r = (float*)malloc(hidden_unit * x_width * sizeof(float));
    hipMemcpy(host_w_r, w_r, hidden_unit * x_width * sizeof(float), hipMemcpyDeviceToHost);
    float* host_u_r = (float*)malloc(hidden_unit * hidden_unit * sizeof(float));
    hipMemcpy(host_u_r, u_r, hidden_unit * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);
    float* host_w_h = (float*)malloc(hidden_unit * x_width * sizeof(float));
    hipMemcpy(host_w_h, w_h, hidden_unit * x_width * sizeof(float), hipMemcpyDeviceToHost);
    float* host_u_h = (float*)malloc(hidden_unit * hidden_unit * sizeof(float));
    hipMemcpy(host_u_h, u_h, hidden_unit * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);
    // end remove
    
    const int threadsPerBlock = 512;
    int blocks = (hidden_unit * batch_size + threadsPerBlock - 1) / threadsPerBlock;

    float* tmp1 = (float*)malloc(hidden_unit * batch_size * sizeof(float));
    float* tmp2 = (float*)malloc(hidden_unit * batch_size * sizeof(float));

    float* device_tmp1;
    float* device_tmp2;
    hipMalloc((void **)&device_tmp1, hidden_unit * batch_size * sizeof(float));
    hipMalloc((void **)&device_tmp1, hidden_unit * batch_size * sizeof(float));

    // z_t = sigmoid(x_t * w_z + old_h_t * u_z + b_z)
    memset(tmp1, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_x_t, host_w_z, tmp1, hidden_unit, batch_size, x_width);
    memset(tmp2, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_old_h_t, host_u_z, tmp2, hidden_unit, batch_size, hidden_unit);
    float* z_t;
    hipMalloc((void **)&z_t, hidden_unit * batch_size * sizeof(float));

    // copy to device
    hipMemcpy(device_tmp1, tmp1, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_tmp2, tmp2, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);

    mat_add_kernel<<<blocks, threadsPerBlock>>>(device_tmp1, device_tmp2, z_t, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(z_t, b_z, z_t, hidden_unit, batch_size);
    mat_sigmoid_kernel<<<blocks, threadsPerBlock>>>(z_t, hidden_unit, batch_size);

    // r_t = sigmoid(x_t * w_r + old_h_t * u_r + b_r)
    memset(tmp1, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_x_t, host_w_r, tmp1, hidden_unit, batch_size, x_width);
    memset(tmp2, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_old_h_t, host_u_r, tmp2, hidden_unit, batch_size, hidden_unit);
    float* r_t;
    hipMalloc((void **)&r_t, hidden_unit * batch_size * sizeof(float));

    // copy to device
    hipMemcpy(device_tmp1, tmp1, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_tmp2, tmp2, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);

    mat_add_kernel<<<blocks, threadsPerBlock>>>(device_tmp1, device_tmp2, r_t, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(r_t, b_r, r_t, hidden_unit, batch_size);
    mat_sigmoid_kernel<<<blocks, threadsPerBlock>>>(r_t, hidden_unit, batch_size);

    // remove after paralleled matrix multiplication
    float* host_r_t = (float*)malloc(batch_size * hidden_unit * sizeof(float));
    hipMemcpy(host_r_t, r_t, batch_size * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);
    // end of remove
    printf("%.f\n", host_r_t[0]);

    // h_hat = phi(x_t * w_h + (r_t . old_h_t) * u_h + b_h)
    memset(tmp1, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_x_t, host_w_h, tmp1, hidden_unit, batch_size, x_width);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(r_t, old_h_t, r_t, hidden_unit, batch_size);
    memset(tmp2, 0, hidden_unit * batch_size * sizeof(float));
    mat_multiplication_kernel(host_r_t, host_u_h, tmp2, hidden_unit, batch_size, hidden_unit);
    float* h_hat;
    hipMalloc((void **)&h_hat, hidden_unit * batch_size * sizeof(float));

    // copy to device
    hipMemcpy(device_tmp1, tmp1, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_tmp2, tmp2, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);

    mat_add_kernel<<<blocks, threadsPerBlock>>>(device_tmp1, device_tmp2, h_hat, hidden_unit, batch_size); 
    mat_add_b_kernel<<<blocks, threadsPerBlock>>>(h_hat, b_h, h_hat, hidden_unit, batch_size);
    mat_tanh_kernel<<<blocks, threadsPerBlock>>>(h_hat, hidden_unit, batch_size);

    // new_h_t = (1-z_t).old_h_t + z_t.h_hat
    float* tmp3;
    hipMalloc((void **)&tmp3, hidden_unit * batch_size * sizeof(float));
    mat_one_sub_kernel<<<blocks, threadsPerBlock>>>(z_t, tmp3, hidden_unit, batch_size);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(tmp3, old_h_t, h_hat, hidden_unit, batch_size);
    mat_hadamard_kernel<<<blocks, threadsPerBlock>>>(z_t, h_hat, h_hat, hidden_unit, batch_size);
    mat_add_kernel<<<blocks, threadsPerBlock>>>(tmp3, h_hat, new_h_t, hidden_unit, batch_size);

}


void
gru_forward_cuda(int batch_size, int x_width, int hidden_unit,
                float* x_t, float* old_h_t, float* new_h_t,
                float* w_z, float* w_r, float* w_h,
                float* u_z, float* u_r, float* u_h,
                float* b_z, float* b_r, float* b_h) {

    float *device_x_t;
    float *device_old_h_t;
    float *device_new_h_t;
    float *device_w_z;
    float *device_w_r;
    float *device_w_h;
    float *device_u_z;
    float *device_u_r;
    float *device_u_h;
    float *device_b_z;
    float *device_b_r;
    float *device_b_h;

    hipMalloc((void **)&device_x_t, batch_size * x_width * sizeof(float));
    hipMalloc((void **)&device_old_h_t, batch_size * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_new_h_t, batch_size * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_z, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_h, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_w_r, x_width * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_z, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_h, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_u_r, hidden_unit * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_z, 1 * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_h, 1 * hidden_unit * sizeof(float));
    hipMalloc((void **)&device_b_r, 1 * hidden_unit * sizeof(float));

    hipMemcpy(device_x_t, x_t, batch_size * x_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_old_h_t, old_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_new_h_t, new_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_z, w_z, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_h, w_h, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_w_r, w_r, x_width * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_z, u_z, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_h, u_h, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_u_r, u_r, hidden_unit * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_z, b_z, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_h, b_h, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b_r, b_r, 1 * hidden_unit * sizeof(float), hipMemcpyHostToDevice);

    gru_forward_kernel(batch_size, x_width, hidden_unit,
                device_x_t, device_old_h_t, device_new_h_t,
                device_w_z, device_w_r, device_w_h,
                device_u_z, device_u_r, device_u_h,
                device_b_z, device_b_r, device_b_h);

    hipDeviceSynchronize();

    hipMemcpy(new_h_t, device_new_h_t, batch_size * hidden_unit * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_x_t);
    hipFree(device_old_h_t);
    hipFree(device_new_h_t);
    hipFree(device_w_z);
    hipFree(device_w_h);
    hipFree(device_w_r);
    hipFree(device_u_z);
    hipFree(device_u_h);
    hipFree(device_u_r);
    hipFree(device_b_z);
    hipFree(device_b_h);
    hipFree(device_b_r);
}


void
print_cuda_info() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
